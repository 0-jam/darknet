#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <time.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "maxpool_layer.h"
#include "cost_layer.h"
#include "normalization_layer.h"
#include "freeweight_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
}

extern "C" float * get_network_output_gpu_layer(network net, int i);
extern "C" float * get_network_delta_gpu_layer(network net, int i);
float *get_network_output_gpu(network net);

void forward_network_gpu(network net, float * input, float * truth, int train)
{
    int i;
    for(i = 0; i < net.n; ++i){
        //clock_t time = clock();
        if(net.types[i] == CONVOLUTIONAL){
            convolutional_layer layer = *(convolutional_layer *)net.layers[i];
            forward_convolutional_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == DECONVOLUTIONAL){
            deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
            forward_deconvolutional_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == COST){
            cost_layer layer = *(cost_layer *)net.layers[i];
            forward_cost_layer_gpu(layer, input, truth);
        }
        else if(net.types[i] == CONNECTED){
            connected_layer layer = *(connected_layer *)net.layers[i];
            forward_connected_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == MAXPOOL){
            maxpool_layer layer = *(maxpool_layer *)net.layers[i];
            forward_maxpool_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == SOFTMAX){
            softmax_layer layer = *(softmax_layer *)net.layers[i];
            forward_softmax_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == DROPOUT){
            if(!train) continue;
            dropout_layer layer = *(dropout_layer *)net.layers[i];
            forward_dropout_layer_gpu(layer, input);
            input = layer.output_gpu;
        }
        else if(net.types[i] == CROP){
            crop_layer layer = *(crop_layer *)net.layers[i];
            forward_crop_layer_gpu(layer, train, input);
            input = layer.output_gpu;
        }
        //hipDeviceSynchronize();
        //printf("Forward %d %s %f\n", i, get_layer_string(net.types[i]), sec(clock() - time));
    }
}

void backward_network_gpu(network net, float * input)
{
    int i;
    float * prev_input;
    float * prev_delta;
    for(i = net.n-1; i >= 0; --i){
        //clock_t time = clock();
        if(i == 0){
            prev_input = input;
            prev_delta = 0;
        }else{
            prev_input = get_network_output_gpu_layer(net, i-1);
            prev_delta = get_network_delta_gpu_layer(net, i-1);
        }
        if(net.types[i] == CONVOLUTIONAL){
            convolutional_layer layer = *(convolutional_layer *)net.layers[i];
            backward_convolutional_layer_gpu(layer, prev_input, prev_delta);
        }
        else if(net.types[i] == DECONVOLUTIONAL){
            deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
            backward_deconvolutional_layer_gpu(layer, prev_input, prev_delta);
        }
        else if(net.types[i] == COST){
            cost_layer layer = *(cost_layer *)net.layers[i];
            backward_cost_layer_gpu(layer, prev_input, prev_delta);
        }
        else if(net.types[i] == CONNECTED){
            connected_layer layer = *(connected_layer *)net.layers[i];
            backward_connected_layer_gpu(layer, prev_input, prev_delta);
        }
        else if(net.types[i] == MAXPOOL){
            maxpool_layer layer = *(maxpool_layer *)net.layers[i];
            backward_maxpool_layer_gpu(layer, prev_delta);
        }
        else if(net.types[i] == DROPOUT){
            dropout_layer layer = *(dropout_layer *)net.layers[i];
            backward_dropout_layer_gpu(layer, prev_delta);
        }
        else if(net.types[i] == SOFTMAX){
            softmax_layer layer = *(softmax_layer *)net.layers[i];
            backward_softmax_layer_gpu(layer, prev_delta);
        }
        //printf("Backward %d %s %f\n", i, get_layer_string(net.types[i]), sec(clock() - time));
    }
}

void update_network_gpu(network net)
{
    int i;
    for(i = 0; i < net.n; ++i){
        if(net.types[i] == CONVOLUTIONAL){
            convolutional_layer layer = *(convolutional_layer *)net.layers[i];
            update_convolutional_layer_gpu(layer);
        }
        else if(net.types[i] == DECONVOLUTIONAL){
            deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
            update_deconvolutional_layer_gpu(layer);
        }
        else if(net.types[i] == CONNECTED){
            connected_layer layer = *(connected_layer *)net.layers[i];
            update_connected_layer_gpu(layer);
        }
    }
}

float * get_network_output_gpu_layer(network net, int i)
{
    if(net.types[i] == CONVOLUTIONAL){
        convolutional_layer layer = *(convolutional_layer *)net.layers[i];
        return layer.output_gpu;
    }
    else if(net.types[i] == DECONVOLUTIONAL){
        deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
        return layer.output_gpu;
    }
    else if(net.types[i] == CONNECTED){
        connected_layer layer = *(connected_layer *)net.layers[i];
        return layer.output_gpu;
    }
    else if(net.types[i] == MAXPOOL){
        maxpool_layer layer = *(maxpool_layer *)net.layers[i];
        return layer.output_gpu;
    }
    else if(net.types[i] == CROP){
        crop_layer layer = *(crop_layer *)net.layers[i];
        return layer.output_gpu;
    }
    else if(net.types[i] == SOFTMAX){
        softmax_layer layer = *(softmax_layer *)net.layers[i];
        return layer.output_gpu;
    } else if(net.types[i] == DROPOUT){
        dropout_layer layer = *(dropout_layer *)net.layers[i];
        return layer.output_gpu;
    }
    return 0;
}

float * get_network_delta_gpu_layer(network net, int i)
{
    if(net.types[i] == CONVOLUTIONAL){
        convolutional_layer layer = *(convolutional_layer *)net.layers[i];
        return layer.delta_gpu;
    }
    else if(net.types[i] == DECONVOLUTIONAL){
        deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
        return layer.delta_gpu;
    }
    else if(net.types[i] == CONNECTED){
        connected_layer layer = *(connected_layer *)net.layers[i];
        return layer.delta_gpu;
    }
    else if(net.types[i] == MAXPOOL){
        maxpool_layer layer = *(maxpool_layer *)net.layers[i];
        return layer.delta_gpu;
    }
    else if(net.types[i] == SOFTMAX){
        softmax_layer layer = *(softmax_layer *)net.layers[i];
        return layer.delta_gpu;
    } else if(net.types[i] == DROPOUT){
        if(i == 0) return 0;
        return get_network_delta_gpu_layer(net, i-1);
    }
    return 0;
}

float train_network_datum_gpu(network net, float *x, float *y)
{
  //clock_t time = clock();
    int x_size = get_network_input_size(net)*net.batch;
    int y_size = get_network_output_size(net)*net.batch;
    if(!*net.input_gpu){
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }else{
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
  //printf("trans %f\n", sec(clock() - time));
  //time = clock();
    forward_network_gpu(net, *net.input_gpu, *net.truth_gpu, 1);
  //printf("forw %f\n", sec(clock() - time));
  //time = clock();
    backward_network_gpu(net, *net.input_gpu);
  //printf("back %f\n", sec(clock() - time));
  //time = clock();
    update_network_gpu(net);
    float error = get_network_cost(net);

    //print_letters(y, 50);
    //float *out = get_network_output_gpu(net);
    //print_letters(out, 50);
  //printf("updt %f\n", sec(clock() - time));
  //time = clock();
    return error;
}

float *get_network_output_layer_gpu(network net, int i)
{
    if(net.types[i] == CONVOLUTIONAL){
        convolutional_layer layer = *(convolutional_layer *)net.layers[i];
        return layer.output;
    }
    else if(net.types[i] == DECONVOLUTIONAL){
        deconvolutional_layer layer = *(deconvolutional_layer *)net.layers[i];
        return layer.output;
    }
    else if(net.types[i] == CONNECTED){
        connected_layer layer = *(connected_layer *)net.layers[i];
        cuda_pull_array(layer.output_gpu, layer.output, layer.outputs*layer.batch);
        return layer.output;
    }
    else if(net.types[i] == MAXPOOL){
        maxpool_layer layer = *(maxpool_layer *)net.layers[i];
        return layer.output;
    }
    else if(net.types[i] == SOFTMAX){
        softmax_layer layer = *(softmax_layer *)net.layers[i];
        pull_softmax_layer_output(layer);
        return layer.output;
    }
    return 0;
}

float *get_network_output_gpu(network net)
{
    int i;
    for(i = net.n-1; i > 0; --i) if(net.types[i] != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(network net, float *input)
{

    int size = get_network_input_size(net) * net.batch;
    float * input_gpu = cuda_make_array(input, size);
    forward_network_gpu(net, input_gpu, 0, 0);
    float *out = get_network_output_gpu(net);
    cuda_free(input_gpu);
    return out;
}

