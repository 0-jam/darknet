#include "hip/hip_runtime.h"
extern "C" {
#include "network.h"
#include "region_layer.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
}

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_swag_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_swag(image im, int num, float thresh, box *boxes, float **probs, char *label);

extern "C" void demo_swag(char *cfgfile, char *weightfile, float thresh)
{
    network net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    region_layer l = net.layers[net.n-1];
    cv::VideoCapture cap(0);

    set_batch_network(&net, 1);
    srand(2222222);
    float nms = .4;
    int j;
    box *boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    float **probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    while(1){
        cv::Mat frame_m;
        cap >> frame_m;
        IplImage frame = frame_m;
        image im = ipl_to_image(&frame);
        rgbgr_image(im);

        image sized = resize_image(im, net.w, net.h);
        float *X = sized.data;
        float *predictions = network_predict(net, X);
        convert_swag_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, thresh, probs, boxes, 0);
        if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
        printf("\033[2J");
        printf("\033[1;1H");
        printf("\nObjects:\n\n");
        draw_swag(im, l.side*l.side*l.n, thresh, boxes, probs, "predictions");

        free_image(im);
        free_image(sized);
        cvWaitKey(1);
    }
}
#else
extern "C" void demo_swag(char *cfgfile, char *weightfile, float thresh){}
#endif

