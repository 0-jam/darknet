extern "C" {
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

extern "C" void forward_deconvolutional_layer_gpu(deconvolutional_layer layer, float *in)
{
    int i;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;

    int m = layer.size*layer.size*layer.n;
    int n = layer.h*layer.w;
    int k = layer.c;

    bias_output_gpu(layer.output_gpu, layer.biases_gpu, layer.batch, layer.n, size);

    for(i = 0; i < layer.batch; ++i){
        float *a = layer.filters_gpu;
        float *b = in + i*layer.c*layer.h*layer.w;
        float *c = layer.col_image_gpu;

        gemm_ongpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_ongpu(c, layer.n, out_h, out_w, layer.size, layer.stride, 0, layer.output_gpu+i*layer.n*size);
    }
    activate_array(layer.output_gpu, layer.batch*layer.n*size, layer.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(deconvolutional_layer layer, float *in, float *delta_gpu)
{
    float alpha = 1./layer.batch;
    int out_h = deconvolutional_out_height(layer);
    int out_w = deconvolutional_out_width(layer);
    int size = out_h*out_w;
    int i;

    gradient_array(layer.output_gpu, size*layer.n*layer.batch, layer.activation, layer.delta_gpu);
    backward_bias(layer.bias_updates_gpu, layer.delta, layer.batch, layer.n, size);

    if(delta_gpu) memset(delta_gpu, 0, layer.batch*layer.h*layer.w*layer.c*sizeof(float));

    for(i = 0; i < layer.batch; ++i){
        int m = layer.c;
        int n = layer.size*layer.size*layer.n;
        int k = layer.h*layer.w;

        float *a = in + i*m*n;
        float *b = layer.col_image_gpu;
        float *c = layer.filter_updates_gpu;

        im2col_ongpu(layer.delta_gpu + i*layer.n*size, layer.n, out_h, out_w, 
                layer.size, layer.stride, 0, b);
        gemm_ongpu(0,1,m,n,k,alpha,a,k,b,k,1,c,n);

        if(delta_gpu){
            int m = layer.c;
            int n = layer.h*layer.w;
            int k = layer.size*layer.size*layer.n;

            float *a = layer.filters_gpu;
            float *b = layer.col_image_gpu;
            float *c = delta_gpu + i*n*m;

            gemm(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
}

extern "C" void pull_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void push_deconvolutional_layer(deconvolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void update_deconvolutional_layer_gpu(deconvolutional_layer layer)
{
    int size = layer.size*layer.size*layer.c*layer.n;

    axpy_ongpu(layer.n, layer.learning_rate, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n,layer.momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(size, -layer.decay, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, layer.learning_rate, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, layer.momentum, layer.filter_updates_gpu, 1);
}

