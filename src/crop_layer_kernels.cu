#include "hip/hip_runtime.h"
extern "C" {
#include "crop_layer.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include "image.h"
}

#define BLOCK 256

__device__ float get_pixel_kernel(float *image, int w, int h, int x, int y, int c)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*(y + c*h)];
}

__device__ float billinear_interpolate_kernel(float *image, int w, int h, float x, float y, int c)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    float dx = x - ix;
    float dy = y - iy;

    float val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy, c) + 
                dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1, c) + 
                (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy, c) +
                dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1, c);
    return val;
}

__global__ void forward_crop_layer_kernel(float *input, int size, int c, int h, int w, int crop_height, int crop_width, int dh, int dw, int flip, float angle, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    float cx = w/2.;
    float cy = h/2.;

    int count = id;
    int j = id % crop_width;
    id /= crop_width;
    int i = id % crop_height;
    id /= crop_height;
    int k = id % c;
    id /= c;
    int b = id;

    input += w*h*c*b;

    int x = (flip) ? w - dw - j - 1 : j + dw;    
    int y = i + dh;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;

    output[count] = billinear_interpolate_kernel(input, w, h, rx, ry, k);
}

extern "C" void forward_crop_layer_gpu(crop_layer layer, network_state state)
{
    int flip = (layer.flip && rand()%2);
    int dh = rand()%(layer.h - layer.crop_height + 1);
    int dw = rand()%(layer.w - layer.crop_width + 1);
    float angle = rand_uniform() - .5;
    if(!state.train){
        angle = 0;
        flip = 0;
        dh = (layer.h - layer.crop_height)/2;
        dw = (layer.w - layer.crop_width)/2;
    }
    int size = layer.batch*layer.c*layer.crop_width*layer.crop_height;

    dim3 dimBlock(BLOCK, 1, 1);
    dim3 dimGrid((size-1)/BLOCK + 1, 1, 1);

    forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK>>>(state.input, size, layer.c, layer.h, layer.w,
                        layer.crop_height, layer.crop_width, dh, dw, flip, angle, layer.output_gpu);
    check_error(hipPeekAtLastError());
}

