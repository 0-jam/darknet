#include "hip/hip_runtime.h"
extern "C" {
#include "crop_layer.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include "image.h"
}

#define BLOCK 256

__device__ float get_pixel_kernel(float *image, int w, int h, int x, int y, int c)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*(y + c*h)];
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y; 
    float b = rgb.z;

    float h, s, v;
    float max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
    float min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
    float delta = max - min;
    v = max;
    if(max == 0){
        s = 0;
        h = -1;
    }else{
        s = delta/max;
        if(r == max){
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y; 
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v*(1-s);
        q = v*(1-s*f);
        t = v*(1-s*(1-f));
        if(index == 0){
            r = v; g = t; b = p;
        } else if(index == 1){
            r = q; g = v; b = p;
        } else if(index == 2){
            r = p; g = v; b = t;
        } else if(index == 3){
            r = p; g = q; b = v;
        } else if(index == 4){
            r = t; g = p; b = v;
        } else {
            r = v; g = p; b = q;
        }
    }
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);
    return make_float3(r, g, b);
}

__device__ float billinear_interpolate_kernel(float *image, int w, int h, float x, float y, int c)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    float dx = x - ix;
    float dy = y - iy;

    float val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy, c) + 
        dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1, c) + 
        (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy, c) +
        dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1, c);
    return val;
}

__global__ void levels_image_kernel(float *image, int batch, int w, int h, float saturation, float exposure, float translate, float scale)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;
    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w*(y + h*2)];
    float g = image[x + w*(y + h*1)];
    float b = image[x + w*(y + h*0)];
    float3 rgb = make_float3(r,g,b);
    float3 hsv = rgb_to_hsv_kernel(rgb);
    hsv.y *= saturation;
    hsv.z *= exposure;
    rgb = hsv_to_rgb_kernel(hsv);
    image[x + w*(y + h*2)] = rgb.x*scale + translate;
    image[x + w*(y + h*1)] = rgb.y*scale + translate;
    image[x + w*(y + h*0)] = rgb.z*scale + translate;
}

__global__ void forward_crop_layer_kernel(float *input, int size, int c, int h, int w, int crop_height, int crop_width, int dh, int dw, int flip, float angle, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    float cx = w/2.;
    float cy = h/2.;

    int count = id;
    int j = id % crop_width;
    id /= crop_width;
    int i = id % crop_height;
    id /= crop_height;
    int k = id % c;
    id /= c;
    int b = id;

    input += w*h*c*b;

    int x = (flip) ? w - dw - j - 1 : j + dw;    
    int y = i + dh;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;

    output[count] = billinear_interpolate_kernel(input, w, h, rx, ry, k);
}

extern "C" void forward_crop_layer_gpu(crop_layer layer, network_state state)
{
    int flip = (layer.flip && rand()%2);
    int dh = rand()%(layer.h - layer.crop_height + 1);
    int dw = rand()%(layer.w - layer.crop_width + 1);
    float radians = layer.angle*3.14159/180.;
    float angle = 2*radians*rand_uniform() - radians;

    float saturation = rand_uniform() + 1;
    if(rand_uniform() > .5) saturation = 1./saturation;
    float exposure = rand_uniform() + 1;
    if(rand_uniform() > .5) exposure = 1./exposure;

    float scale = 2;
    float translate = -1;

    if(!state.train){
        angle = 0;
        flip = 0;
        dh = (layer.h - layer.crop_height)/2;
        dw = (layer.w - layer.crop_width)/2;
        saturation = 1;
        exposure = 1;
    }

    int size = layer.batch * layer.w * layer.h;

    levels_image_kernel<<<cuda_gridsize(size), BLOCK>>>(state.input, layer.batch, layer.w, layer.h, saturation, exposure, translate, scale);
    check_error(hipPeekAtLastError());
    
    size = layer.batch*layer.c*layer.crop_width*layer.crop_height;

    forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK>>>(state.input, size, layer.c, layer.h, layer.w,
            layer.crop_height, layer.crop_width, dh, dw, flip, angle, layer.output_gpu);
    check_error(hipPeekAtLastError());

/*
       cuda_pull_array(layer.output_gpu, layer.output, size);
       image im = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 0*(size/layer.batch));
       image im2 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 1*(size/layer.batch));
       image im3 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 2*(size/layer.batch));
       show_image(im, "cropped");
       show_image(im2, "cropped2");
       show_image(im3, "cropped3");
       cvWaitKey(0);
       */
}

