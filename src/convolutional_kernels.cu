#include "hip/hip_runtime.h"
extern "C" {
#include "convolutional_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void bias_output_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] = biases[filter];
}

extern "C" void bias_output_gpu(float *output, float *biases, int batch, int n, int size)
{
    dim3 dimBlock(BLOCK, 1, 1);
    dim3 dimGrid((size-1)/BLOCK + 1, n, batch);

    bias_output_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
    check_error(hipPeekAtLastError());
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size, float scale)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if(p == 0){
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += scale * part[i];
    }
}

extern "C" void backward_bias_gpu(float *bias_updates, float *delta, int batch, int n, int size)
{
    float alpha = 1./batch;

    backward_bias_kernel<<<n, BLOCK>>>(bias_updates, delta, batch, n, size, alpha);
    check_error(hipPeekAtLastError());
}

extern "C" void forward_convolutional_layer_gpu(convolutional_layer layer, float *in)
{
    int i;
    int m = layer.n;
    int k = layer.size*layer.size*layer.c;
    int n = convolutional_out_height(layer)*
        convolutional_out_width(layer);

    bias_output_gpu(layer.output_gpu, layer.biases_gpu, layer.batch, layer.n, n);

    for(i = 0; i < layer.batch; ++i){
        im2col_ongpu(in, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, layer.col_image_gpu);
        float * a = layer.filters_gpu;
        float * b = layer.col_image_gpu;
        float * c = layer.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
    activate_array_ongpu(layer.output_gpu, m*n*layer.batch, layer.activation);
}

extern "C" void backward_convolutional_layer_gpu(convolutional_layer layer, float *in, float *delta_gpu)
{
    float alpha = 1./layer.batch;
    int i;
    int m = layer.n;
    int n = layer.size*layer.size*layer.c;
    int k = convolutional_out_height(layer)*
        convolutional_out_width(layer);

    gradient_array_ongpu(layer.output_gpu, m*k*layer.batch, layer.activation, layer.delta_gpu);
    backward_bias_gpu(layer.bias_updates_gpu, layer.delta_gpu, layer.batch, layer.n, k);

    if(delta_gpu) scal_ongpu(layer.batch*layer.h*layer.w*layer.c, 0, delta_gpu, 1);

    for(i = 0; i < layer.batch; ++i){
        float * a = layer.delta_gpu;
        float * b = layer.col_image_gpu;
        float * c = layer.filter_updates_gpu;

        im2col_ongpu(in, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, layer.col_image_gpu);
        gemm_ongpu(0,1,m,n,k,alpha,a + i*m*k,k,b,k,1,c,n);

        if(delta_gpu){

            float * a = layer.filters_gpu;
            float * b = layer.delta_gpu;
            float * c = layer.col_image_gpu;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(layer.col_image_gpu, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, delta_gpu);
        }
    }
}

extern "C" void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void update_convolutional_layer_gpu(convolutional_layer layer)
{
    int size = layer.size*layer.size*layer.c*layer.n;

/*
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, size);
    cuda_pull_array(layer.filters_gpu, layer.filters, size);
    printf("Filter: %f updates: %f\n", mag_array(layer.filters, size), layer.learning_rate*mag_array(layer.filter_updates, size));
    */

    axpy_ongpu(layer.n, layer.learning_rate, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n,layer.momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(size, -layer.decay, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, layer.learning_rate, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, layer.momentum, layer.filter_updates_gpu, 1);
    //pull_convolutional_layer(layer);
}

