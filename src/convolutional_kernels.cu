#include "hip/hip_runtime.h"
extern "C" {
#include "convolutional_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void bias(int n, int size, float *biases, float *output)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] = biases[filter];
}

extern "C" void bias_output_gpu(const convolutional_layer layer)
{
    int size = convolutional_out_height(layer)*convolutional_out_width(layer);

    dim3 dimBlock(BLOCK, 1, 1);
    dim3 dimGrid((size-1)/BLOCK + 1, layer.n, layer.batch);

    bias<<<dimGrid, dimBlock>>>(layer.n, size, layer.biases_gpu, layer.output_gpu);
    check_error(hipPeekAtLastError());
}

__global__ void learn_bias(int batch, int n, int size, float *delta, float *bias_updates, float scale)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if(p == 0){
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += scale * part[i];
    }
}

extern "C" void learn_bias_convolutional_layer_ongpu(convolutional_layer layer)
{
    int size = convolutional_out_height(layer)*convolutional_out_width(layer);
    float alpha = 1./layer.batch;

    learn_bias<<<layer.n, BLOCK>>>(layer.batch, layer.n, size, layer.delta_gpu, layer.bias_updates_gpu, alpha);
    check_error(hipPeekAtLastError());
}

extern "C" void test_learn_bias(convolutional_layer l)
{
    int i;
    int size = convolutional_out_height(l) * convolutional_out_width(l);
    for(i = 0; i < size*l.batch*l.n; ++i){
        l.delta[i] = rand_uniform();
    }
    for(i = 0; i < l.n; ++i){
        l.bias_updates[i] = rand_uniform();
    }
    cuda_push_array(l.delta_gpu, l.delta, size*l.batch*l.n);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    float *gpu = (float *) calloc(l.n, sizeof(float));
    cuda_pull_array(l.bias_updates_gpu, gpu, l.n);
    for(i = 0; i < l.n; ++i) printf("%.9g %.9g\n", l.bias_updates[i], gpu[i]);
    learn_bias_convolutional_layer_ongpu(l);
    learn_bias_convolutional_layer(l);
    cuda_pull_array(l.bias_updates_gpu, gpu, l.n);
    for(i = 0; i < l.n; ++i) printf("%.9g %.9g\n", l.bias_updates[i], gpu[i]);
}

extern "C" void forward_convolutional_layer_gpu(convolutional_layer layer, float *in)
{
    int i;
    int m = layer.n;
    int k = layer.size*layer.size*layer.c;
    int n = convolutional_out_height(layer)*
        convolutional_out_width(layer);

    bias_output_gpu(layer);

    for(i = 0; i < layer.batch; ++i){
        im2col_ongpu(in, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, layer.col_image_gpu);
        float * a = layer.filters_gpu;
        float * b = layer.col_image_gpu;
        float * c = layer.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
    activate_array_ongpu(layer.output_gpu, m*n*layer.batch, layer.activation);
}

extern "C" void backward_convolutional_layer_gpu(convolutional_layer layer, float *in, float *delta_gpu)
{
    float alpha = 1./layer.batch;
    int i;
    int m = layer.n;
    int n = layer.size*layer.size*layer.c;
    int k = convolutional_out_height(layer)*
        convolutional_out_width(layer);
    gradient_array_ongpu(layer.output_gpu, m*k*layer.batch, layer.activation, layer.delta_gpu);
    learn_bias_convolutional_layer_ongpu(layer);

    if(delta_gpu) scal_ongpu(layer.batch*layer.h*layer.w*layer.c, 0, delta_gpu, 1);

    for(i = 0; i < layer.batch; ++i){
        float * a = layer.delta_gpu;
        float * b = layer.col_image_gpu;
        float * c = layer.filter_updates_gpu;

        im2col_ongpu(in, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, layer.col_image_gpu);
        gemm_ongpu(0,1,m,n,k,alpha,a + i*m*k,k,b,k,1,c,n);

        if(delta_gpu){

            float * a = layer.filters_gpu;
            float * b = layer.delta_gpu;
            float * c = layer.col_image_gpu;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(layer.col_image_gpu, i*layer.c*layer.h*layer.w, layer.c,  layer.h,  layer.w,  layer.size,  layer.stride, layer.pad, delta_gpu);
        }
    }
}

extern "C" void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.filters_gpu, layer.filters, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.filter_updates_gpu, layer.filter_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
}

extern "C" void update_convolutional_layer_gpu(convolutional_layer layer)
{
    int size = layer.size*layer.size*layer.c*layer.n;

/*
    cuda_pull_array(layer.filter_updates_gpu, layer.filter_updates, size);
    cuda_pull_array(layer.filters_gpu, layer.filters, size);
    printf("Filter: %f updates: %f\n", mag_array(layer.filters, size), layer.learning_rate*mag_array(layer.filter_updates, size));
    */

    axpy_ongpu(layer.n, layer.learning_rate, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n,layer.momentum, layer.bias_updates_gpu, 1);

    axpy_ongpu(size, -layer.decay, layer.filters_gpu, 1, layer.filter_updates_gpu, 1);
    axpy_ongpu(size, layer.learning_rate, layer.filter_updates_gpu, 1, layer.filters_gpu, 1);
    scal_ongpu(size, layer.momentum, layer.filter_updates_gpu, 1);
    //pull_convolutional_layer(layer);
}

