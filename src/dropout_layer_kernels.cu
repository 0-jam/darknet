#include "hip/hip_runtime.h"
extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) output[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

extern "C" void forward_dropout_layer_gpu(dropout_layer layer, float * input)
{
    int j;
    int size = layer.inputs*layer.batch;
    for(j = 0; j < size; ++j) layer.rand[j] = rand_uniform();
    cuda_push_array(layer.rand_gpu, layer.rand, layer.inputs*layer.batch);

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(input, size, layer.rand_gpu, layer.probability,
            layer.scale, layer.output_gpu);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_dropout_layer_gpu(dropout_layer layer, float *delta)
{
    if(!delta) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(delta, size, layer.rand_gpu, layer.probability,
            layer.scale, delta);
    check_error(hipPeekAtLastError());
}
