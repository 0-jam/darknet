#include "hip/hip_runtime.h"
extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
#include "params.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

extern "C" void forward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if (!state.train) return;
    int j;
    int size = layer.inputs*layer.batch;
    for(j = 0; j < size; ++j) layer.rand[j] = rand_uniform();
    cuda_push_array(layer.rand_gpu, layer.rand, layer.inputs*layer.batch);

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.input, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_dropout_layer_gpu(dropout_layer layer, network_state state)
{
    if(!state.delta) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(state.delta, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}
